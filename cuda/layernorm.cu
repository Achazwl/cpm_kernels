#include "hip/hip_runtime.h"
#include "reduce.cuh"
#include <hip/hip_fp16.h>
#include "common.h"

// block <batch_idx, m/32>   thread <32, 32>
CPM_KERNEL_EXPORT void cu_layernorm_forward(
    int32_t batch, int32_t n, int32_t m,
    const half *mat,    // (batch, n, m)
    half *out,          // (batch, n, m)
    float eps,
    bool rd_mean
) {
    int32_t base_mat_idx = (blockIdx.x * n + threadIdx.y) * m + blockIdx.y * WARP_SZ + threadIdx.x;
    int32_t col_idx = blockIdx.y * WARP_SZ + threadIdx.x;

    float local_total_v = 0.0;
    float local_total_v2 = 0.0;

    // nested reduce loops to achieve higher precision
    for (int i = 0; i < n; i += WARP_SZ * WARP_SZ) {
        
        // inner loop reduce
        float inner_v = 0.0;
        float inner_v2 = 0.0;
        for (int j = 0; j < WARP_SZ * WARP_SZ && i + j < n; j += WARP_SZ) {
            // reduce per block
            float v = 0, tv = 0, tv2 = 0;
            if (col_idx < m && i + j + threadIdx.y < n) {
                v = (float)mat[base_mat_idx + (i + j) * m];
            }
            if (rd_mean) {
                tv = transposeReduceSum(v);
            }
            tv2 = transposeReduceSum(v * v);
            if (threadIdx.y * WARP_SZ == j) {
                inner_v = tv;
                inner_v2 = tv2;
            }
        }

        // outter loop reduce, reduce per 32 blocks
        if (rd_mean) {
            local_total_v += transposeReduceSum(inner_v);
        }
        local_total_v2 += transposeReduceSum(inner_v2);
    }

    local_total_v2 = local_total_v2 / (float)n;
    if (rd_mean) {
        local_total_v = local_total_v / (float)n;
        local_total_v2 -= local_total_v * local_total_v;
    }

    local_total_v2 = rsqrtf(local_total_v2 + eps);

    float local_mean =  local_total_v;
    float local_var = local_total_v2;
    if (rd_mean) {
        for (int i = 0; i < n; i += WARP_SZ) {
            if (col_idx < m && i + threadIdx.y < n) {
                out[base_mat_idx + i * m] = __float2half(((float)mat[base_mat_idx + i * m] - local_mean) * local_var);
            }
        }
    } else {
        for (int i = 0; i < n; i += blockDim.y) {
            if (col_idx < m && i + threadIdx.y < n) {
                out[base_mat_idx + i * m] = __float2half((float)mat[base_mat_idx + i * m] * local_var);
            }
        }
    }
}

// block <batch_idx, m/32>   thread <32, 32>
// WARNING: this function is the same as cu_layernorm_forward now.
CPM_KERNEL_EXPORT void cu_layernorm_inplace_forward(
    int32_t batch, int32_t n, int32_t m,
    half *mat,    // (batch, n, m)
    float eps,
    bool rd_mean
) {
    int32_t base_mat_idx = (blockIdx.x * n + threadIdx.y) * m + blockIdx.y * WARP_SZ + threadIdx.x;
    int32_t col_idx = blockIdx.y * WARP_SZ + threadIdx.x;

    float local_total_v = 0.0;
    float local_total_v2 = 0.0;

    // nested reduce loops to achieve higher precision
    for (int i = 0; i < n; i += WARP_SZ * WARP_SZ) {
        
        // inner loop reduce
        float inner_v = 0.0;
        float inner_v2 = 0.0;
        for (int j = 0; j < WARP_SZ * WARP_SZ && i + j < n; j += WARP_SZ) {
            // reduce per block
            float v = 0, tv = 0, tv2 = 0;
            if (col_idx < m && i + j + threadIdx.y < n) {
                v = (float)mat[base_mat_idx + (i + j) * m];
            }
            if (rd_mean) {
                tv = transposeReduceSum(v);
            }
            tv2 = transposeReduceSum(v * v);
            if (threadIdx.y * WARP_SZ == j) {
                inner_v = tv;
                inner_v2 = tv2;
            }
        }

        // outter loop reduce, reduce per 32 blocks
        if (rd_mean) {
            local_total_v += transposeReduceSum(inner_v);
        }
        local_total_v2 += transposeReduceSum(inner_v2);
    }

    local_total_v2 = local_total_v2 / (float)n;
    if (rd_mean) {
        local_total_v = local_total_v / (float)n;
        local_total_v2 -= local_total_v * local_total_v;
    }

    local_total_v2 = rsqrtf(local_total_v2 + eps);

    float local_mean =  local_total_v;
    float local_var = local_total_v2;
    if (rd_mean) {
        for (int i = 0; i < n; i += WARP_SZ) {
            if (col_idx < m && i + threadIdx.y < n) {
                mat[base_mat_idx + i * m] = __float2half(((float)(mat[base_mat_idx + i * m]) - local_mean) * local_var);
            }
        }
    } else {
        for (int i = 0; i < n; i += blockDim.y) {
            if (col_idx < m && i + threadIdx.y < n) {
                mat[base_mat_idx + i * m] = __float2half((float)(mat[base_mat_idx + i * m]) * local_var);
            }
        }
    }
}

// block <batch_idx, offset_m/32>   thread <32, 32>
CPM_KERNEL_EXPORT void cu_layernorm_forward_v(
    int32_t batch, int32_t n, int32_t m,
    const half *mat,    // (batch, n, m)
    half *out,          // (batch, n, m)
    half *out_var,      // (batch, m)
    float eps
) {
    int32_t base_mat_idx = (blockIdx.x * n + threadIdx.y) * m + blockIdx.y * WARP_SZ + threadIdx.x;
    int32_t col_idx = blockIdx.y * WARP_SZ + threadIdx.x;

    float local_total_v2 = 0.0;

    // nested reduce loops to achieve higher precision
    for (int i = 0; i < n; i += WARP_SZ * WARP_SZ) {
        
        // inner loop reduce
        float inner_v2 = 0.0;
        for (int j = 0; j < WARP_SZ * WARP_SZ && i + j < n; j += WARP_SZ) {
            // reduce per block
            float v = 0, tv2 = 0;
            if (col_idx < m && i + j + threadIdx.y < n) {
                v = (float)mat[base_mat_idx + (i + j) * m];
            }
            tv2 = transposeReduceSum(v * v);
            if (threadIdx.y * WARP_SZ == j) {
                inner_v2 = tv2;
            }
        }

        // outter loop reduce, reduce per 32 blocks
        local_total_v2 += transposeReduceSum(inner_v2);
    }

    float local_var = rsqrtf(local_total_v2 / (float)n + eps);

    if (threadIdx.y == 0 && col_idx < m) out_var[blockIdx.x * m + col_idx] = __float2half(local_var);

    for (int i = 0; i < n; i += blockDim.y) {
        if (col_idx < m && i + threadIdx.y < n) {
            out[base_mat_idx + i * m] = __float2half((float)mat[base_mat_idx + i * m] * local_var);
        }
    }
}

// block <batch_idx, offset_m/32>   thread <32, 32>
CPM_KERNEL_EXPORT void cu_layernorm_forward_mv(
    int32_t batch, int32_t n, int32_t m,
    const half *mat,    // (batch, n, m)
    half *out,
    half *out_mean,
    half *out_var,
    float eps
) {
    int32_t base_mat_idx = (blockIdx.x * n + threadIdx.y) * m + blockIdx.y * WARP_SZ + threadIdx.x;
    int32_t col_idx = blockIdx.y * WARP_SZ + threadIdx.x;

    float local_total_v = 0.0;
    float local_total_v2 = 0.0;

    // nested reduce loops to achieve higher precision
    for (int i = 0; i < n; i += WARP_SZ * WARP_SZ) {
        
        // inner loop reduce
        float inner_v = 0.0;
        float inner_v2 = 0.0;
        for (int j = 0; j < WARP_SZ * WARP_SZ && i + j < n; j += WARP_SZ) {
            // reduce per block
            float v = 0, tv = 0, tv2 = 0;
            if (col_idx < m && i + j + threadIdx.y < n) {
                v = (float)mat[base_mat_idx + (i + j) * m];
            }
            tv = transposeReduceSum(v);
            tv2 = transposeReduceSum(v * v);
            if (threadIdx.y * WARP_SZ == j) {
                inner_v = tv;
                inner_v2 = tv2;
            }
        }

        // outter loop reduce, reduce per 32 blocks
        local_total_v += transposeReduceSum(inner_v);
        local_total_v2 += transposeReduceSum(inner_v2);
    }

    local_total_v = local_total_v / (float)n;
    local_total_v2 = rsqrtf(local_total_v2 / (float)n - local_total_v * local_total_v + eps);

    if (threadIdx.y == 0 && col_idx < m) {
        out_var[blockIdx.x * m + col_idx] = __float2half(local_total_v2);
        out_mean[blockIdx.x * m + col_idx] = __float2half(local_total_v);
    }

    float local_mean =  local_total_v;
    float local_var = local_total_v2;
    for (int i = 0; i < n; i += WARP_SZ) {
        if (col_idx < m && i + threadIdx.y < n) {
            out[base_mat_idx + i * m] = __float2half(((float)(mat[base_mat_idx + i * m]) - local_mean) * local_var);
        }
    }
}

// block <batch_idx, offset_m/32>   thread <32, 32>
CPM_KERNEL_EXPORT void cu_layernorm_backward_v(
    int32_t batch, int32_t n, int32_t m,
    const half *mat,        // (batch, n, m)
    const half *grad_in,    // (batch, n, m)
    const half *var,        // (batch, m) 
    half *grad_out
) {
    int32_t base_mat_idx = (blockIdx.x * n + threadIdx.y) * m + blockIdx.y * WARP_SZ + threadIdx.x;
    int32_t col_idx = blockIdx.y * WARP_SZ + threadIdx.x;

    float local_grad_var = 0;

    float local_var = col_idx < m ? (float)__ldg(var + blockIdx.x * m + col_idx) : 0.0;
    float n_half_rsqrt_v3 = -0.5 * local_var * local_var * local_var;

    for (int i = 0; i < n; i += WARP_SZ * WARP_SZ) {
        float inner_v = 0;
        for (int j = 0; j < WARP_SZ * WARP_SZ && i + j < n; j += WARP_SZ) {
            float v = 0;
            if (col_idx < m && i + j + threadIdx.y < n) {
                v = (float)grad_in[base_mat_idx + (i + j) * m] * 
                                    n_half_rsqrt_v3 * 
                                    ((float)mat[base_mat_idx + (i + j) * m]);
            }
            v = transposeReduceSum(v);
            if (threadIdx.y * WARP_SZ == j) inner_v = v;
        }
        local_grad_var += transposeReduceSum(inner_v);
    }

    for (int i = 0; i < n; i += WARP_SZ) {
        if (col_idx < m && i + threadIdx.y < n) {
            grad_out[base_mat_idx + i * m] = __float2half(
                (float)grad_in[base_mat_idx + i * m] * local_var +
                ((local_grad_var * (float)mat[base_mat_idx + i * m] * 2) / (float)n)
            );
        }
    }
}

// block <batch_idx, offset_m/32>   thread <32, 32>
CPM_KERNEL_EXPORT void cu_layernorm_backward_mv(
    int32_t batch, int32_t n, int32_t m,
    const half *mat,        // (batch, n, m)
    const half *grad_in,    // (batch, n, m)
    const half *mean,       // (batch, m)
    const half *var,        // (batch, m) 
    half *grad_out
) {
    int32_t base_mat_idx = (blockIdx.x * n + threadIdx.y) * m + blockIdx.y * WARP_SZ + threadIdx.x;
    int32_t col_idx = blockIdx.y * WARP_SZ + threadIdx.x;


    float local_grad_var = 0;
    float local_grad_mean = 0;

    float local_mean =  col_idx < m ? (float)__ldg(mean + blockIdx.x * m + col_idx) : 0.0;
    float local_var = col_idx < m ? (float)__ldg(var + blockIdx.x * m + col_idx) : 0.0;

    float n_half_rsqrt_v3 = -0.5 * local_var * local_var * local_var;


    for (int i = 0; i < n; i += WARP_SZ * WARP_SZ) {
        float inner_m = 0;
        float inner_v = 0;

        for (int j = 0; j < WARP_SZ * WARP_SZ && i + j < n; j += WARP_SZ) {
            float vv = 0, vm = 0;
            if (col_idx < m && i + j + threadIdx.y < n) {
                float gi = (float)grad_in[base_mat_idx + (i + j) * m];
                vv = gi * n_half_rsqrt_v3 * ((float)mat[base_mat_idx + (i + j) * m] - local_mean);
                vm = -gi * local_var;
            }
            vv = transposeReduceSum(vv);
            vm = transposeReduceSum(vm);
            if (threadIdx.y * WARP_SZ == j) {
                inner_v = vv;
                inner_m = vm;
            }
        }

        local_grad_var += transposeReduceSum(inner_v);
        local_grad_mean += transposeReduceSum(inner_m);
    }

    local_grad_mean -= 2 * local_grad_var * local_mean;

    for (int i = 0; i < n; i += WARP_SZ) {
        if (col_idx < m && i + threadIdx.y < n) {
            grad_out[base_mat_idx + i * m] = __float2half(
                (float)grad_in[base_mat_idx + i * m] * local_var +
                ((local_grad_mean + local_grad_var * (float)mat[base_mat_idx + i * m] * 2) / (float)n)
            );
        }
    }
}


// block <batch>    thread <min(round_up(n, 32), 1024)>
CPM_KERNEL_EXPORT void cu_layernorm_step(
    int32_t batch, int32_t n,
    const half *mat,    // (batch, n)
    half *out,          // (batch, n)
    float eps,
    bool rd_mean
) {
    int32_t base_mat_idx = blockIdx.x * n;

    float local_total_v = 0;
    float local_total_v2 = 0;
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        float v =  mat[base_mat_idx + i];
        if (rd_mean) local_total_v += v;
        local_total_v2 += v * v;
    }

    __shared__ float global_mean;
    __shared__ float global_var;

    if (rd_mean) local_total_v = blockReduceSum(local_total_v);
    local_total_v2 = blockReduceSum(local_total_v2);
    if (threadIdx.x == 0) {
        if (rd_mean) {
            global_mean = local_total_v / (float)n;
            global_var = local_total_v2 / (float)n - global_mean * global_mean;
        } else {
            global_var = local_total_v2 / (float)n;
            global_mean = 0;
        }
    }
    __syncthreads();
    local_total_v2 = rsqrtf(global_var + eps);
    local_total_v = global_mean;
    
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        out[base_mat_idx + i] = __float2half((__half2float(mat[base_mat_idx + i]) - local_total_v) * local_total_v2);
    }
}

// block <batch>    thread <min(round_up(n, 32), 1024)>
CPM_KERNEL_EXPORT void cu_layernorm_step_inplace(
    int32_t batch, int32_t n,
    half *mat,    // (batch, n)
    float eps,
    bool rd_mean
) {
    int32_t base_mat_idx = blockIdx.x * n;

    float local_total_v = 0;
    float local_total_v2 = 0;
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        float v = mat[base_mat_idx + i];
        if (rd_mean) local_total_v += v;
        local_total_v2 += v * v;
    }

    __shared__ float global_mean;
    __shared__ float global_var;

    if (rd_mean) local_total_v = blockReduceSum(local_total_v);
    local_total_v2 = blockReduceSum(local_total_v2);
    if (threadIdx.x == 0) {
        if (rd_mean) {
            global_mean = local_total_v / (float)n;
            global_var = local_total_v2 / (float)n - global_mean * global_mean;
        } else {
            global_var = local_total_v2 / (float)n;
            global_mean = 0;
        }
    }
    __syncthreads();
    local_total_v2 = rsqrtf(global_var + eps);
    local_total_v = global_mean;
    
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        mat[base_mat_idx + i] = __float2half((__half2float(mat[base_mat_idx + i]) - local_total_v) * local_total_v2);
    }
}