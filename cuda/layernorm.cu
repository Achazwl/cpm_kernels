#include "hip/hip_runtime.h"
#include "reduce.cuh"
#include <hip/hip_fp16.h>
#include "common.h"

// block <batch_idx, m/32>   thread <32, 32>
CPM_KERNEL_EXPORT void cu_layernorm_forward(
    int32_t batch, int32_t n, int32_t m,
    const half *mat,    // (batch, n, m)
    half *out,          // (batch, n, m)
    float eps,
    bool rd_mean
) {
    int32_t base_mat_idx = (blockIdx.x * n + threadIdx.y) * m + blockIdx.y * WARP_SZ + threadIdx.x;
    int32_t col_idx = blockIdx.y * WARP_SZ + threadIdx.x;

    float local_total_v = 0.0;
    float local_total_v2 = 0.0;
    for (int i = 0; i < n; i += WARP_SZ) {
        float v = 0;
        if (col_idx < m && i + threadIdx.y < n) {
            v = (float)__ldg(mat + base_mat_idx + i * m);
        }

        if (rd_mean) local_total_v += v;
        local_total_v2 += v * v;
    }

    local_total_v2 = transposeReduceSum(local_total_v2) / (float)n;
    if (rd_mean) {
        local_total_v = transposeReduceSum(local_total_v) / (float)n;
        local_total_v2 -= local_total_v * local_total_v;
    }

    local_total_v2 = rsqrtf(local_total_v2 + eps);

    float local_mean =  local_total_v;
    float local_var = local_total_v2;
    if (rd_mean) {
        for (int i = 0; i < n; i += WARP_SZ) {
            if (col_idx < m && i + threadIdx.y < n) {
                out[base_mat_idx + i * m] = __float2half(((float)__ldg(mat + base_mat_idx + i * m) - local_mean) * local_var);
            }
        }
    } else {
        for (int i = 0; i < n; i += blockDim.y) {
            if (col_idx < m && i + threadIdx.y < n) {
                out[base_mat_idx + i * m] = __float2half((float)__ldg(mat + base_mat_idx + i * m) * local_var);
            }
        }
    }
}

// block <batch_idx, offset_m/32>   thread <32, 32>
CPM_KERNEL_EXPORT void cu_layernorm_forward_v(
    int32_t batch, int32_t n, int32_t m,
    const half *mat,    // (batch, n, m)
    half *out,          // (batch, n, m)
    half *out_var,      // (batch, m)
    float eps
) {
    int32_t base_mat_idx = (blockIdx.x * n + threadIdx.y) * m + blockIdx.y * WARP_SZ + threadIdx.x;
    int32_t col_idx = blockIdx.y * WARP_SZ + threadIdx.x;

    float local_total_v2 = 0.0;
    for (int i = 0; i < n; i += WARP_SZ) {
        float v = 0;
        if (col_idx < m && i + threadIdx.y < n) {
            v = (float)__ldg(mat + base_mat_idx + i * m);
        }
        local_total_v2 += v * v;
    }
    float local_var = rsqrtf(transposeReduceSum(local_total_v2) / (float)n + eps);

    if (threadIdx.y == 0 && col_idx < m) out_var[blockIdx.x * m + col_idx] = __float2half(local_var);

    for (int i = 0; i < n; i += blockDim.y) {
        if (col_idx < m && i + threadIdx.y < n) {
            out[base_mat_idx + i * m] = __float2half((float)__ldg(mat + base_mat_idx + i * m) * local_var);
        }
    }
}

// block <batch_idx, offset_m/32>   thread <32, 32>
CPM_KERNEL_EXPORT void cu_layernorm_forward_mv(
    int32_t batch, int32_t n, int32_t m,
    const half *mat,    // (batch, n, m)
    half *out,
    half *out_mean,
    half *out_var,
    float eps
) {
    int32_t base_mat_idx = (blockIdx.x * n + threadIdx.y) * m + blockIdx.y * WARP_SZ + threadIdx.x;
    int32_t col_idx = blockIdx.y * WARP_SZ + threadIdx.x;

    float local_total_v = 0.0;
    float local_total_v2 = 0.0;
    for (int i = 0; i < n; i += WARP_SZ) {
        float v = 0;
        if (col_idx < m && i + threadIdx.y < n) {
            v = (float)__ldg(mat + base_mat_idx + i * m);
        }
        local_total_v += v;
        local_total_v2 += v * v;
    }

    local_total_v = transposeReduceSum(local_total_v) / (float)n;
    local_total_v2 = rsqrtf(transposeReduceSum(local_total_v2) / (float)n - local_total_v * local_total_v + eps);

    if (threadIdx.y == 0 && col_idx < m) {
        out_var[blockIdx.x * m + col_idx] = __float2half(local_total_v2);
        out_mean[blockIdx.x * m + col_idx] = __float2half(local_total_v);
    }

    float local_mean =  local_total_v;
    float local_var = local_total_v2;
    for (int i = 0; i < n; i += WARP_SZ) {
        if (col_idx < m && i + threadIdx.y < n) {
            out[base_mat_idx + i * m] = __float2half(((float)(mat[base_mat_idx + i * m]) - local_mean) * local_var);
        }
    }
}

// block <batch_idx, offset_m/32>   thread <32, 32>
CPM_KERNEL_EXPORT void cu_layernorm_backward_v(
    int32_t batch, int32_t n, int32_t m,
    const half *mat,        // (batch, n, m)
    const half *grad_in,    // (batch, n, m)
    const half *var,        // (batch, m) 
    half *grad_out
) {
    int32_t base_mat_idx = (blockIdx.x * n + threadIdx.y) * m + blockIdx.y * WARP_SZ + threadIdx.x;
    int32_t col_idx = blockIdx.y * WARP_SZ + threadIdx.x;

    float local_grad_var = 0;

    float local_var = col_idx < m ? (float)__ldg(var + blockIdx.x * m + col_idx) : 0.0;
    float n_half_rsqrt_v3 = -0.5 * local_var * local_var * local_var;

    for (int i = 0; i < n; i += WARP_SZ) {
        if (col_idx < m && i + threadIdx.y < n) {
            local_grad_var += (float)__ldg(grad_in + base_mat_idx + i * m) * n_half_rsqrt_v3 * ((float)__ldg(mat + base_mat_idx + i * m));
        }
    }

    local_grad_var = transposeReduceSum(local_grad_var);

    for (int i = 0; i < n; i += WARP_SZ) {
        if (col_idx < m && i + threadIdx.y < n) {
            grad_out[base_mat_idx + i * m] = __float2half(
                (float)__ldg(grad_in + base_mat_idx + i * m) * local_var +
                ((local_grad_var * (float)__ldg(mat + base_mat_idx + i * m) * 2) / (float)n)
            );
        }
    }
}

// block <batch_idx, offset_m/32>   thread <32, 32>
CPM_KERNEL_EXPORT void cu_layernorm_backward_mv(
    int32_t batch, int32_t n, int32_t m,
    const half *mat,        // (batch, n, m)
    const half *grad_in,    // (batch, n, m)
    const half *mean,       // (batch, m)
    const half *var,        // (batch, m) 
    half *grad_out
) {
    int32_t base_mat_idx = (blockIdx.x * n + threadIdx.y) * m + blockIdx.y * WARP_SZ + threadIdx.x;
    int32_t col_idx = blockIdx.y * WARP_SZ + threadIdx.x;


    float local_grad_var = 0;
    float local_grad_mean = 0;

    float local_mean =  col_idx < m ? (float)__ldg(mean + blockIdx.x * m + col_idx) : 0.0;
    float local_var = col_idx < m ? (float)__ldg(var + blockIdx.x * m + col_idx) : 0.0;

    float n_half_rsqrt_v3 = -0.5 * local_var * local_var * local_var;


    for (int i = 0; i < n; i += WARP_SZ) {
        if (col_idx < m && i + threadIdx.y < n) {
            float gi = (float)__ldg(grad_in + base_mat_idx + i * m);
            local_grad_var += gi * n_half_rsqrt_v3 * ((float)__ldg(mat + base_mat_idx + i * m) - local_mean);
            local_grad_mean += -gi * local_var;
        }
    }

    local_grad_var = transposeReduceSum(local_grad_var);
    local_grad_mean = transposeReduceSum(local_grad_mean);

    local_grad_mean -= 2 * local_grad_var * local_mean;

    for (int i = 0; i < n; i += WARP_SZ) {
        if (col_idx < m && i + threadIdx.y < n) {
            grad_out[base_mat_idx + i * m] = __float2half(
                (float)__ldg(grad_in + base_mat_idx + i * m) * local_var +
                ((local_grad_mean + local_grad_var * (float)__ldg(mat + base_mat_idx + i * m) * 2) / (float)n)
            );
        }
    }
}
